#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
#include <sys/time.h>
}

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_yolo_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_yolo(image im, int num, float thresh, box *boxes, float **probs);

extern "C" char *voc_names[];
extern "C" image voc_labels[];

static float **probs;
static box *boxes;
static network net;
static image in   ;
static image in_s ;
static image det  ;
static image det_s;
static image disp ;
static cv::VideoCapture cap;
static float fps = 0;
static float demo_thresh = 0;

// new stuff
#include "feature_matcher.h"
#include <vector>
#include <string>
#include <deque>
std::vector<std::deque<cv::Mat> > person_db;
int pool_size = 2;
int wait_period = 1;
int since_last = 0;
std::vector<int> active;
cv::Mat current_img;
std::vector<cv::Mat> image_matches;
std::vector<cv::Mat> bad_matches;
std::vector<int> indices_matches;
bool no_match;
IplImage* im_ptr = NULL;
int frame_num = 0;

void *fetch_in_thread(void *ptr)
{
    cv::Mat frame_m;
    cap >> frame_m;
    IplImage frame = frame_m;
    in = ipl_to_image(&frame);
    rgbgr_image(in);
    in_s = resize_image(in, net.w, net.h);
    ++frame_num;
    return 0;
}

void image_to_mat(image p, cv::Mat& m) {
    int x,y,k;
    image copy = copy_image(p);
    constrain_image(copy);
    if(p.c == 3) rgbgr_image(copy);
    //normalize_image(copy);

    // char buff[256];
    // //sprintf(buff, "%s (%d)", name, windows);
    // sprintf(buff, "%s", name);

    m.create(p.h, p.w, CV_8UC3);

    // IplImage *disp = cvCreateImage(cvSize(p.w,p.h), IPL_DEPTH_8U, p.c);
    // int step = disp->widthStep;
    // cvNamedWindow(buff, CV_WINDOW_NORMAL); 
    //cvMoveWindow(buff, 100*(windows%10) + 200*(windows/10), 100*(windows%10));
    // ++windows;
    for(y = 0; y < p.h; ++y){
        for(x = 0; x < p.w; ++x){
            for(k= 0; k < p.c; ++k){
                m.at<cv::Vec3b>(y,x)[k] = (unsigned char)(get_pixel(copy,x,y,k)*255);
                // m.at<uchar>(y, x, 0) = 255; //(unsigned char)(get_pixel(copy,x,y,k)*255);
                // m.at<uchar>(y, x, 1) = 0; //(unsigned char)(get_pixel(copy,x,y,k)*255);
                // m.at<uchar>(y, x, 2) = 0; //(unsigned char)(get_pixel(copy,x,y,k)*255);
                // disp->imageData[y*step + x*p.c + k] = (unsigned char)(get_pixel(copy,x,y,k)*255);
            }
        }
    }
    free_image(copy);

    // m = cv::Mat(disp);
    // return disp;
}

void track_person(image image_im, int num, float thresh, box *boxes, float **probs, char **names, image *labels, int classes)
{
    int cls_person = 14;

    active = std::vector<int>(person_db.size());
    std::vector<cv::Rect> rects;
    std::vector<int> person_ids;
    image_matches.clear();
    indices_matches.clear();
    bad_matches.clear();

    image_to_mat(image_im, current_img);

    for(int i = 0; i < num; ++i){
        int cls = max_index(probs[i], classes);
        float prob = probs[i][cls];
        if(cls == cls_person && prob > thresh){

            box& b = boxes[i];
            int left  = (b.x-b.w/2.)*image_im.w;
            int right = (b.x+b.w/2.)*image_im.w;
            int top   = (b.y-b.h/2.)*image_im.h;
            int bot   = (b.y+b.h/2.)*image_im.h;

            if(left < 0) left = 0;
            if(right > image_im.w-1) right = image_im.w-1;
            if(top < 0) top = 0;
            if(bot > image_im.h-1) bot = image_im.h-1;


            cv::Rect rect(left, top, right-left, bot-top);
            cv::Mat new_box = current_img(rect);

            int found = -1;
            int max_person = -1;
            int max_matches = 0;
            cv::Mat max_image_match;

            // search match between current person with person database
            for (int j = 0, len = person_db.size(); j < len; ++j) {
                if (active[j] == 0) {
                    int vote = 0;
                    cv::Mat image_match;
                    for (int k = 0, len = person_db[j].size(); k < len; ++k) {
                        int match_result = matchFeatures(person_db[j][k], new_box, image_match);

                        if (match_result > 0) {
                            vote++;
                        }  else {
                            bad_matches.push_back(image_match);
                        }
                    }
                    if (vote >= person_db[j].size()/2) {
                        max_person = j;
                        max_matches = vote;
                        max_image_match = image_match;
                        break;
                    }
                }
            }

            // found person, update old person portfolio 
            if (max_person != -1) {
                found = max_person;
                if (since_last < wait_period) {
                    ++since_last;
                } else {
                    person_db[max_person].push_back(new_box.clone());
                    since_last = 0;
                }
                active[max_person] = max_matches;
                image_matches.push_back(max_image_match);
                indices_matches.push_back(max_person+1);
            }

            // did not find any person, creating a new profile in person database
            if (found == -1) {
                if (person_db.size() == 0) {
                    found = person_db.size();
                    person_db.push_back(std::deque<cv::Mat>());
                    person_db.back().push_back(new_box.clone());
                    active.push_back(1);
                }
            }

            if (found != -1) {
                rects.push_back(rect);
                person_ids.push_back(found+1);
                if (person_db[found].size() > pool_size) {
                    person_db[found].pop_front();
                }
            } else {
                rects.push_back(rect);
                person_ids.push_back(0);
            }
        }
    }

    for (int i = 0, len = person_ids.size(); i < len; ++i) {
        // label person
        char person_callname[50];
        sprintf(person_callname, "Person %d", person_ids[i]);
        if (person_ids[i])
            printf("Person %d, matches %d\n", person_ids[i], active[person_ids[i]-1]);
        cv::putText(current_img, person_callname, cv::Point(rects[i].x+10, rects[i].y+30), cv::FONT_HERSHEY_SIMPLEX, 1, cv::Scalar( 255,0,0 ), 2);
        cv::rectangle(current_img, rects[i], cv::Scalar(240,128,128), 3);
    }

    if (image_matches.size()) {
        printf("Match accepted!\n");
        no_match = false;
    } else {
        printf("Match rejected or no match!\n");
        no_match = true;
    }
}

void *detect_in_thread(void *ptr)
{
    float nms = .4;

    detection_layer l = net.layers[net.n-1];
    float *X = det_s.data;
    float *predictions = network_predict(net, X);
    free_image(det_s);
    convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
    if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
    // printf("\033[2J");
    // printf("\033[1;1H");
    printf("\nFPS:%.0f\n",fps);
    printf("Objects:\n\n");

    // new stuff
    track_person(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, 20);
    // draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, 20);

    return 0;
}

extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index)
{
    demo_thresh = thresh;
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);

    srand(2222222);


    bool use_video = false;
    if (use_video) {
        // Open video file
        std::string video_path = "drone.mp4";
        cv::VideoCapture vid(video_path);
        cap = vid;
        if(!cap.isOpened()) error(("Couldn't open video: " + video_path + "\n").c_str());
    } else {
        // Open camera
        cv::VideoCapture cam(cam_index);
        cap = cam;
        if(!cap.isOpened()) error("Couldn't connect to webcam.\n");
    }

    detection_layer l = net.layers[net.n-1];
    int j;

    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));

    pthread_t fetch_thread;
    pthread_t detect_thread;

    fetch_in_thread(0);
    det = in;
    det_s = in_s;

    fetch_in_thread(0);
    detect_in_thread(0);
    disp = det;
    det = in;
    det_s = in_s;

    int fast_forward = 1;

    while(1){
        struct timeval tval_before, tval_after, tval_result;
        gettimeofday(&tval_before, NULL);
        if(pthread_create(&fetch_thread, 0, fetch_in_thread, 0)) error("Thread creation failed");
        if(pthread_create(&detect_thread, 0, detect_in_thread, 0)) error("Thread creation failed");
        pthread_join(fetch_thread, 0);
        pthread_join(detect_thread, 0);

        // if (person_db.size()) {
        //     for (int i = 0, len = person_db[0].size(); i < len; ++i) {
        //         if (person_db[0][i].rows) {
        //             char match_name[50];
        //             sprintf(match_name, "Person 1 Sample %d", i);
        //             cv::imshow(match_name, person_db[0][i]);
        //             char key = cv::waitKey(1);
        //             if (key == 's') {
        //                 fast_forward = 0;
        //             }
        //         }
        //     }
            
        // }
        for (int i = 0, len = image_matches.size(); i < len; ++i) {
            if (image_matches[i].rows) {
                char match_name[50];
                sprintf(match_name, "Match %d", indices_matches[i]);
                cv::imshow(match_name, image_matches[i]);
                char key = cv::waitKey(1);
                if (key == 's') {
                    fast_forward = 0;
                }
            }
        }
        // for (int i = 0, len = bad_matches.size(); i < len; ++i) {
        //     if (bad_matches[i].rows) {
        //         char match_name[50];
        //         sprintf(match_name, "Bad Match %d", i);
        //         cv::imshow(match_name, bad_matches[i]);
        //         char key = cv::waitKey(1);
        //         if (key == 's') {
        //             fast_forward = 0;
        //         }
        //     }
        // }
        if (current_img.rows) {
            cv::imshow("YOLO", current_img);
            char key = cv::waitKey(1);
            if (key == 's') {
                fast_forward = 0;
            }
        }

        printf("Frame: %d\n", frame_num);

        if (fast_forward == 0) {
            char key = cv::waitKey(0);
            if (key == 'f') {
                fast_forward = 1;
            } else if (key == 's') {
                fast_forward = 0;
            }
        }
        

        // show_image(disp, "YOLO");
        free_image(disp);
        cvWaitKey(1);

        disp  = det;
        det   = in;
        det_s = in_s;

        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        float curr = 1000000.f/((long int)tval_result.tv_usec);
        fps = .9*fps + .1*curr;
    }
}
#else
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index){
    fprintf(stderr, "YOLO demo needs OpenCV for webcam images.\n");
}
#endif

